#include "hip/hip_runtime.h"
//============================================================================
// Name        : rcuda.cu
// Author      : Jan Kyska
// Version     : 1.00
// Description : Generator of FreeRainbowTables / MD5, MD4, NTLM, SHA1, LM
//============================================================================ 

#include <stdio.h>  
#include <hip/hip_runtime.h>  
#include "rcuda.h"

#define   GRID_X_L2     6
#define   GRID_Y_L2     6
#define   BLOCK_X_L2    6
#define   GRID_X_SIZE   (1<<GRID_X_L2)
#define   GRID_Y_SIZE   (1<<GRID_Y_L2)
#define   BLOCK_X_SIZE  (1<<BLOCK_X_L2)
#define   PLAIN_MAX_SIZE     20
#define   KERN_CHAIN_SIZE   100
#define   CHAR_SET_MAXLEN   512
#define   SHIDX(x)      ((x)<<4)


__device__ uint64 *dataHeap;
__device__ unsigned char *plStart;
__device__ uint3 *plDimVec;
__device__ unsigned char *plChrSet;
__device__ int *plCpPos;
__device__ int plCpPosSize;
__device__ uint64 reduceOffset;
__device__ uint64 plainSpaceTotal;
__device__ uint64 rPlainSpaceTotal;

#define RTGEN_PROLOGUE  \
	unsigned int *hData;  \
	unsigned int idx;  \
	uint64 idx64;  \
	unsigned int nPos, ii;  \
	unsigned int cpcheck, checkpoint;  \
	\
	__shared__ unsigned int shData[SHIDX(BLOCK_X_SIZE)];  \
	__shared__ unsigned char cplChrSet[CHAR_SET_MAXLEN];  \
	__shared__ unsigned char cplStart[PLAIN_MAX_SIZE];  \
	__shared__ uint3 cplDimVec[PLAIN_MAX_SIZE];  \
	\
	if(threadIdx.x == 0) {  \
		nPos = ((((blockIdx.y<<GRID_X_L2) + blockIdx.x)<<BLOCK_X_L2) + threadIdx.x) << 1;  \
		for(ii = 0; ii < BLOCK_X_SIZE; ii++, nPos+=2) {  \
			hData = shData + ((ii>>4)<<8)+(ii&15);  \
			hData[SHIDX(0)] = dataHeap[nPos];  \
			hData[SHIDX(1)] = dataHeap[nPos]>>32;  \
			hData[SHIDX(2)] = dataHeap[nPos+1];  \
		}  \
		memcpy(cplChrSet, plChrSet, CHAR_SET_MAXLEN);  \
		memcpy(cplStart, plStart, PLAIN_MAX_SIZE);  \
		memcpy(cplDimVec, plDimVec, PLAIN_MAX_SIZE*sizeof(uint3));  \
	}  \
	__syncthreads();  \
	\
	hData = shData + ((threadIdx.x>>4)<<8)+(threadIdx.x&15);  \
	\
	idx64 = hData[SHIDX(1)];  \
	idx64 = (idx64<<32) | hData[SHIDX(0)];  \
	cpcheck = hData[SHIDX(2)];  \
	checkpoint = cpcheck&0x0000ffff;  \
	cpcheck = cpcheck>>16;  \
	\
	for(nPos = chainStart; nPos < chainStop; nPos++) {


#define RTGEN_EPILOGUE  \
		idx64 = hData[SHIDX(1)];  \
		idx64 = (idx64<<32) | hData[SHIDX(0)];  \
		idx64 += reduceOffset + nPos;  \
		uiDiv64 = __umul64hi(idx64, rPlainSpaceTotal);  \
		idx64 -= uiDiv64*plainSpaceTotal;  \
		if(idx64 >= plainSpaceTotal)  \
			idx64 -= plainSpaceTotal;  \
		\
		if(cpcheck < plCpPosSize && nPos == plCpPos[cpcheck]) {  \
			checkpoint |= ((unsigned int)idx64&1) << cpcheck;  \
			cpcheck++;  \
		}  \
	}  \
	\
	hData[SHIDX(0)] = idx64;  \
	hData[SHIDX(1)] = idx64>>32;  \
	hData[SHIDX(2)] = (cpcheck<<16)|(checkpoint&0x0000ffff);  \
	__syncthreads();  \
	\
	if(threadIdx.x == 0) {  \
		nPos = ((((blockIdx.y<<GRID_X_L2) + blockIdx.x)<<BLOCK_X_L2) + threadIdx.x) << 1;  \
		for(ii = 0; ii < BLOCK_X_SIZE; ii++, nPos+=2) {  \
			hData = shData + ((ii>>4)<<8)+(ii&15);  \
			dataHeap[nPos] = ((uint64)hData[SHIDX(1)]<<32)|(uint64)hData[SHIDX(0)];  \
			dataHeap[nPos+1] = hData[SHIDX(2)];  \
		}  \
	}


#include "rcuda_md5.inc"
#include "rcuda_md4.inc"
#include "rcuda_ntlm.inc"
#include "rcuda_sha1.inc"
#include "rcuda_lm.inc"

extern "C" int CalcChainsOnCUDA(const rcuda::RCudaTask* task, uint64 *resultBuff) {
	hipError_t cuErr;
	char buff[PLAIN_MAX_SIZE];
	uint64 *data;
	unsigned char *stPlain;
	uint3 *dimVec;
	unsigned char *charSet;
	int *cpPos;
	uint64 uiVal64;

	if(task->charSetSize > CHAR_SET_MAXLEN)
		return -1;

	switch(task->hash) {
	case rcuda::RHASH_MD5:
	case rcuda::RHASH_MD4:
	case rcuda::RHASH_NTLM:
	case rcuda::RHASH_SHA1:
	case rcuda::RHASH_LM:
		break;
	default:	
		return 0;
	}
	
	memset(buff, 0, PLAIN_MAX_SIZE);
	hipMalloc((void**)&data, task->idxCount*2*sizeof(uint64));
	hipMalloc((void**)&stPlain, PLAIN_MAX_SIZE);
	hipMalloc((void**)&dimVec, max(task->dimVecSize, PLAIN_MAX_SIZE)*sizeof(uint3));
	hipMalloc((void**)&charSet, CHAR_SET_MAXLEN);
	hipMalloc((void**)&cpPos, task->cpPosSize*sizeof(int));

	hipMemcpy(data, resultBuff, task->idxCount*2*sizeof(uint64), hipMemcpyHostToDevice);
	hipMemcpy(stPlain, buff, PLAIN_MAX_SIZE, hipMemcpyHostToDevice);
	hipMemcpy(stPlain, task->stPlain, min(task->stPlainSize, PLAIN_MAX_SIZE), hipMemcpyHostToDevice);
	hipMemcpy(dimVec, task->dimVec, min(task->dimVecSize, PLAIN_MAX_SIZE)*sizeof(uint3), hipMemcpyHostToDevice);
	hipMemcpy(charSet, task->charSet, min(task->charSetSize, CHAR_SET_MAXLEN), hipMemcpyHostToDevice);
	hipMemcpy(cpPos, task->cpPositions, task->cpPosSize*sizeof(int), hipMemcpyHostToDevice);

	hipMemcpyToSymbol(HIP_SYMBOL(dataHeap), &data, sizeof(data));
	hipMemcpyToSymbol(HIP_SYMBOL(plStart), &stPlain, sizeof(stPlain));
	hipMemcpyToSymbol(HIP_SYMBOL(plDimVec), &dimVec, sizeof(dimVec));
	hipMemcpyToSymbol(HIP_SYMBOL(plChrSet), &charSet, sizeof(charSet));
	hipMemcpyToSymbol(HIP_SYMBOL(plCpPos), &cpPos, sizeof(cpPos));
	hipMemcpyToSymbol(HIP_SYMBOL(plCpPosSize), &task->cpPosSize, sizeof(task->cpPosSize));
	hipMemcpyToSymbol(HIP_SYMBOL(reduceOffset), &task->reduceOffset, sizeof(task->reduceOffset));
	hipMemcpyToSymbol(HIP_SYMBOL(plainSpaceTotal), &task->plainSpaceTotal, sizeof(task->plainSpaceTotal));
	uiVal64 = (uint64)-1/task->plainSpaceTotal;
	hipMemcpyToSymbol(HIP_SYMBOL(rPlainSpaceTotal), &uiVal64, sizeof(uiVal64));

	int grSizeX = (task->idxCount-1)/BLOCK_X_SIZE + 1;
	int grSizeY = (grSizeX-1)/GRID_X_SIZE + 1;
	grSizeX = GRID_X_SIZE;
	dim3 numBlocks(grSizeX, grSizeY);
	cuErr = hipSuccess;
	
	hipSetDeviceFlags(hipDeviceScheduleBlockingSync);

	for(int idx = 0; idx < task->rainbowChainLen-1 && cuErr == hipSuccess; idx+=KERN_CHAIN_SIZE) {
		switch(task->hash) {
		case rcuda::RHASH_MD5:
			RTGenMD5Kernel<<<numBlocks, BLOCK_X_SIZE>>>(idx, min(idx+KERN_CHAIN_SIZE, task->rainbowChainLen-1));
			break;
		case rcuda::RHASH_MD4:
			RTGenMD4Kernel<<<numBlocks, BLOCK_X_SIZE>>>(idx, min(idx+KERN_CHAIN_SIZE, task->rainbowChainLen-1));
			break;
		case rcuda::RHASH_NTLM:
			RTGenNTLMKernel<<<numBlocks, BLOCK_X_SIZE>>>(idx, min(idx+KERN_CHAIN_SIZE, task->rainbowChainLen-1));
			break;
		case rcuda::RHASH_SHA1:
			RTGenSHA1Kernel<<<numBlocks, BLOCK_X_SIZE>>>(idx, min(idx+KERN_CHAIN_SIZE, task->rainbowChainLen-1));
			break;
		case rcuda::RHASH_LM:
			RTGenLMKernel<<<numBlocks, BLOCK_X_SIZE>>>(idx, min(idx+KERN_CHAIN_SIZE, task->rainbowChainLen-1));
			break;
		}
		cuErr = hipGetLastError();
		if(cuErr == hipSuccess)
			cuErr = hipDeviceSynchronize();
	}

	if(cuErr == hipSuccess)
		hipMemcpy(resultBuff, data, task->idxCount*2*sizeof(uint64), hipMemcpyDeviceToHost);
	else
		fprintf(stderr, "Error happened: %d (%s)\n", cuErr, hipGetErrorString(cuErr));	

	hipFree(cpPos);
	hipFree(charSet);
	hipFree(dimVec);
	hipFree(stPlain);
	hipFree(data);
	return cuErr==hipSuccess? task->idxCount : -1;
}

extern "C" int GetChainsBufferSize(int minSize) {
	int grSizeX = (minSize-1)/BLOCK_X_SIZE + 1;
	int grSizeY = (grSizeX-1)/GRID_X_SIZE + 1;
	grSizeX = GRID_X_SIZE;
	return grSizeX*grSizeY*BLOCK_X_SIZE;
}

extern "C" int SetCudaDevice(int device) {
	return hipSetDevice(device);
}
