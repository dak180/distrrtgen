#include "hip/hip_runtime.h"
#include <stdio.h>  
#include <hip/hip_runtime.h>  
#include "rcuda.h"

#define   GRID_X_L2     6
#define   GRID_Y_L2     6
#define   BLOCK_X_L2    7
#define   GRID_X_SIZE   (1<<GRID_X_L2)
#define   GRID_Y_SIZE   (1<<GRID_Y_L2)
#define   BLOCK_X_SIZE  (1<<BLOCK_X_L2)
#define   KERN_CHAIN_SIZE   100
#define   SHIDX(x)      ((x)<<4)


__device__ __constant__ unsigned int h[4] = { 0x67452301, 0xEFCDAB89, 0x98BADCFE, 0x10325476 };
__device__ __constant__ unsigned char r[64] = { \
							7, 12, 17, 22,  7, 12, 17, 22,  7, 12, 17, 22,  7, 12, 17, 22, \
							5,  9, 14, 20,  5,  9, 14, 20,  5,  9, 14, 20,  5,  9, 14, 20, \
							4, 11, 16, 23,  4, 11, 16, 23,  4, 11, 16, 23,  4, 11, 16, 23, \
							6, 10, 15, 21,  6, 10, 15, 21,  6, 10, 15, 21,  6, 10, 15, 21 };
__device__ __constant__ unsigned char g[64] = { \
							0, 1,  2,  3,   4,  5,  6,  7,   8,  9, 10, 11,  12, 13, 14, 15, \
							1, 6, 11,  0,   5, 10, 15,  4,   9, 14,  3,  8,  13,  2,  7, 12, \
							5, 8, 11, 14,   1,  4,  7, 10,  13,  0,  3,  6,   9, 12, 15,  2, \
							0, 7, 14,  5,  12,  3, 10,  1,   8, 15,  6, 13,   4, 11,  2,  9 };
__device__ __constant__ unsigned int ac[64] = { \
							0xd76aa478, 0xe8c7b756, 0x242070db, 0xc1bdceee, \
							0xf57c0faf, 0x4787c62a, 0xa8304613, 0xfd469501, \
							0x698098d8, 0x8b44f7af, 0xffff5bb1, 0x895cd7be, \
							0x6b901122, 0xfd987193, 0xa679438e, 0x49b40821, \
							\
							0xf61e2562, 0xc040b340, 0x265e5a51, 0xe9b6c7aa, \
							0xd62f105d, 0x02441453, 0xd8a1e681, 0xe7d3fbc8, \
							0x21e1cde6, 0xc33707d6, 0xf4d50d87, 0x455a14ed, \
							0xa9e3e905, 0xfcefa3f8, 0x676f02d9, 0x8d2a4c8a, \
							\
							0xfffa3942, 0x8771f681, 0x6d9d6122, 0xfde5380c, \
							0xa4beea44, 0x4bdecfa9, 0xf6bb4b60, 0xbebfbc70, \
							0x289b7ec6, 0xeaa127fa, 0xd4ef3085, 0x04881d05, \
							0xd9d4d039, 0xe6db99e5, 0x1fa27cf8, 0xc4ac5665, \
							\
							0xf4292244, 0x432aff97, 0xab9423a7, 0xfc93a039, \
							0x655b59c3, 0x8f0ccc92, 0xffeff47d, 0x85845dd1, \
							0x6fa87e4f, 0xfe2ce6e0, 0xa3014314, 0x4e0811a1, \
							0xf7537e82, 0xbd3af235, 0x2ad7d2bb, 0xeb86d391 };

__device__ __constant__ unsigned int testData[16] = { \
							0x79706d63, 0x6d627667, 0x00000080, 0x00000000, \
							0x00000000, 0x00000000, 0x00000000, 0x00000000, \
							0x00000000, 0x00000000, 0x00000000, 0x00000000, \
							0x00000000, 0x00000000, 0x00000040, 0x00000000 };

__device__ unsigned int FF(unsigned int a, unsigned int b, unsigned int c, unsigned int d, int i, const unsigned int* data) {
	unsigned int ret;
	ret = a + ((b&c)|((~b)&d)) + ac[i] + data[SHIDX(g[i])];
	ret = (ret<<r[i])|(ret>>(32-r[i]));
	ret += b;
	return ret;
}

__device__ unsigned int GG(unsigned int a, unsigned int b, unsigned int c, unsigned int d, int i, const unsigned int* data) {
	unsigned int ret;
	ret = a + ((b&d)|(c&(~d))) + ac[i] + data[SHIDX(g[i])];
	ret = (ret<<r[i])|(ret>>(32-r[i]));
	ret += b;
	return ret;
}

__device__ unsigned int HH(unsigned int a, unsigned int b, unsigned int c, unsigned int d, int i, const unsigned int* data) {
	unsigned int ret;
	ret = a + (b^c^d) + ac[i] + data[SHIDX(g[i])];
	ret = (ret<<r[i])|(ret>>(32-r[i]));
	ret += b;
	return ret;
}

__device__ unsigned int II(unsigned int a, unsigned int b, unsigned int c, unsigned int d, int i, const unsigned int* data) {
	unsigned int ret;
	ret = a + (c^(b|(~d))) + ac[i] + data[SHIDX(g[i])];
	ret = (ret<<r[i])|(ret>>(32-r[i]));
	ret += b;
	return ret;
}

__device__ void MD5(unsigned int* dataHash) {
	unsigned int a = h[0], b = h[1], c = h[2], d = h[3], x;
	int ii;

	// Round 1
	for(ii = 0; ii < 16; ii++) {
		x = b;
		b = FF(a, b, c, d, ii, dataHash);
		a = d; d = c; c = x;
	}

	// Round 2
	for(; ii < 32; ii++) {
		x = b;
		b = GG(a, b, c, d, ii, dataHash);
		a = d; d = c; c = x;
	}
	
	// Round 3
	for(; ii < 48; ii++) {
		x = b;
		b = HH(a, b, c, d, ii, dataHash);
		a = d; d = c; c = x;
	}
	
	// Round 4
	for(; ii < 64; ii++) {
		x = b;
		b = II(a, b, c, d, ii, dataHash);
		a = d; d = c; c = x;
	}

	dataHash[SHIDX(0)] = a + h[0];
	dataHash[SHIDX(1)] = b + h[1];
	dataHash[SHIDX(2)] = c + h[2];
	dataHash[SHIDX(3)] = d + h[3];
}


#define   PLAIN_MAX_SIZE     20

__device__ unsigned __int64 *dataHeap;
__device__ unsigned char *plStart;
__device__ uint3 *plDimVec;
__device__ unsigned char *plChrSet;
__device__ int *plCpPos;
__device__ int plCpPosSize;
__device__ unsigned __int64 reduceOffset;
__device__ unsigned __int64 plainSpaceTotal;
__device__ unsigned __int64 rPlainSpaceTotal;


__global__ void RTGenMD5Kernel(unsigned int chainStart, unsigned int chainStop) {
	unsigned int *hData;
	uint3 dimItem;
	unsigned int uiVal, uiMul, uiDiv, idx;
	unsigned __int64 uiVal64, uiMul64, uiDiv64, idx64;
	unsigned int nPos, size, ii, jj, kk;
	unsigned int cpcheck, checkpoint;
	unsigned int plain;

	__shared__ unsigned int shData[SHIDX(BLOCK_X_SIZE)];

	if(threadIdx.x == 0) {
		nPos = ((((blockIdx.y<<GRID_X_L2) + blockIdx.x)<<BLOCK_X_L2) + threadIdx.x) << 1;
		for(ii = 0; ii < BLOCK_X_SIZE; ii++, nPos+=2) {
			hData = shData + ((ii>>4)<<8)+(ii&15);
			hData[SHIDX(0)] = dataHeap[nPos];
			hData[SHIDX(1)] = dataHeap[nPos]>>32;
			hData[SHIDX(2)] = dataHeap[nPos+1];
		}
	}
	__syncthreads();

	hData = shData + ((threadIdx.x>>4)<<8)+(threadIdx.x&15);
	
	idx64 = hData[SHIDX(1)];
	idx64 = (idx64<<32) | hData[SHIDX(0)];
	cpcheck = hData[SHIDX(2)];
	checkpoint = cpcheck&0x0000ffff;
	cpcheck = cpcheck>>16;

	for(nPos = chainStart; nPos < chainStop; nPos++) {
		// transform to the plain text
		plain = 0x80;
		jj = (PLAIN_MAX_SIZE>>2)+1;
		for(ii = 0; idx64 > 0xfffffff0ull && ii < PLAIN_MAX_SIZE; ii++) {
			uiVal64 = idx64 + plStart[ii];
			uiVal64--;
			dimItem = plDimVec[ii];
			uiMul64 = (unsigned __int64)dimItem.y<<32;
			idx64 = __umul64hi(uiVal64, uiMul64);
			uiDiv64 = uiVal64 - idx64*(unsigned __int64)dimItem.x;
			uiVal = __umulhi((unsigned int)uiDiv64, dimItem.y);
			uiDiv = (unsigned int)uiDiv64 - uiVal * dimItem.x;
			idx64 += uiVal;
			if(uiDiv >= dimItem.x) {
				uiDiv -= dimItem.x;
				idx64++;
			}
			plain = (plain<<8) | plChrSet[dimItem.z + uiDiv];
			if((ii&3) == 2) {
				hData[SHIDX(jj--)] = plain;
				plain = 0;
			}
		}
		for(idx = (unsigned int)idx64; idx != 0 && ii < PLAIN_MAX_SIZE; ii++) {
			uiVal = idx + plStart[ii];
			uiVal--;
			dimItem = plDimVec[ii];
			idx = __umulhi(uiVal, dimItem.y);
			uiDiv = uiVal - idx*dimItem.x;
			if(uiDiv >= dimItem.x) {
				uiDiv -= dimItem.x;
				idx++;
			}
			plain = (plain<<8) | plChrSet[dimItem.z + uiDiv];
			if((ii&3) == 2) {
				hData[SHIDX(jj--)] = plain;
				plain = 0;
			}
		}

		// prepare for MD5
		size = ii;
		ii = ((((3-(ii&3))<<3)-1)&0x1f)+1;
		plain = plain<<ii;
		for(jj++, kk = 0; jj <= (PLAIN_MAX_SIZE>>2)+1; plain = hData[SHIDX(jj++)], kk++)
			hData[SHIDX(kk)] = (plain>>ii)|(hData[SHIDX(jj)]<<(32-ii));
		hData[SHIDX(kk)] = plain>>ii;
		for(kk++; kk < 14; kk++)
			hData[SHIDX(kk)] = 0;
		hData[SHIDX(kk++)] = size<<3;
		hData[SHIDX(kk)] = 0;

		// hash
		MD5(hData);

		idx64 = hData[SHIDX(1)];
		idx64 = (idx64<<32) | hData[SHIDX(0)];
		idx64 += reduceOffset + nPos;
		uiDiv64 = __umul64hi(idx64, rPlainSpaceTotal);
		idx64 -= uiDiv64*plainSpaceTotal;
		if(idx64 >= plainSpaceTotal)
			idx64 -= plainSpaceTotal;
			
		if(cpcheck < plCpPosSize && nPos == plCpPos[cpcheck]) {
			checkpoint |= ((unsigned int)idx64&1) << cpcheck;
			cpcheck++;
		}
	}

	hData[SHIDX(0)] = idx64;
	hData[SHIDX(1)] = idx64>>32;
	hData[SHIDX(2)] = (cpcheck<<16)|(checkpoint&0x0000ffff);
	__syncthreads();
	
	if(threadIdx.x == 0) {
		nPos = ((((blockIdx.y<<GRID_X_L2) + blockIdx.x)<<BLOCK_X_L2) + threadIdx.x) << 1;
		for(ii = 0; ii < BLOCK_X_SIZE; ii++, nPos+=2) {
			hData = shData + ((ii>>4)<<8)+(ii&15);
			dataHeap[nPos] = ((unsigned __int64)hData[SHIDX(1)]<<32)|(unsigned __int64)hData[SHIDX(0)];
			dataHeap[nPos+1] = hData[SHIDX(2)];
		}
	}
	__syncthreads();
}


extern "C" int CalcChainsOnCUDA(const rcuda::RCudaTask* task, unsigned __int64 *resultBuff) {
	hipError_t cuErr;
	char buff[PLAIN_MAX_SIZE];
	unsigned __int64 *data;
	unsigned char *stPlain;
	uint3 *dimVec;
	unsigned char *charSet;
	int *cpPos;
	unsigned __int64 uiVal64;
	time_t tStart, tEnd;
	if(task->hash != rcuda::RHASH_MD5)
		return 0;

	memset(buff, 0, PLAIN_MAX_SIZE);
	hipMalloc((void**)&data, task->idxCount*2*sizeof(unsigned __int64));
	hipMalloc((void**)&stPlain, PLAIN_MAX_SIZE);
	hipMalloc((void**)&dimVec, task->dimVecSize*sizeof(uint3));
	hipMalloc((void**)&charSet, task->charSetSize);
	hipMalloc((void**)&cpPos, task->cpPosSize*sizeof(int));

	hipMemcpy(data, resultBuff, task->idxCount*2*sizeof(unsigned __int64), hipMemcpyHostToDevice);
	hipMemcpy(stPlain, buff, PLAIN_MAX_SIZE, hipMemcpyHostToDevice);
	hipMemcpy(stPlain, task->stPlain, min(task->stPlainSize, PLAIN_MAX_SIZE), hipMemcpyHostToDevice);
	hipMemcpy(dimVec, task->dimVec, task->dimVecSize*sizeof(uint3), hipMemcpyHostToDevice);
	hipMemcpy(charSet, task->charSet, task->charSetSize, hipMemcpyHostToDevice);
	hipMemcpy(cpPos, task->cpPositions, task->cpPosSize*sizeof(int), hipMemcpyHostToDevice);

	hipMemcpyToSymbol(HIP_SYMBOL(dataHeap), &data, sizeof(data));
	hipMemcpyToSymbol(HIP_SYMBOL(plStart), &stPlain, sizeof(stPlain));
	hipMemcpyToSymbol(HIP_SYMBOL(plDimVec), &dimVec, sizeof(dimVec));
	hipMemcpyToSymbol(HIP_SYMBOL(plChrSet), &charSet, sizeof(charSet));
	hipMemcpyToSymbol(HIP_SYMBOL(plCpPos), &cpPos, sizeof(cpPos));
	hipMemcpyToSymbol(HIP_SYMBOL(plCpPosSize), &task->cpPosSize, sizeof(task->cpPosSize));
	hipMemcpyToSymbol(HIP_SYMBOL(reduceOffset), &task->reduceOffset, sizeof(task->reduceOffset));
	hipMemcpyToSymbol(HIP_SYMBOL(plainSpaceTotal), &task->plainSpaceTotal, sizeof(task->plainSpaceTotal));
	uiVal64 = (unsigned __int64)-1/task->plainSpaceTotal;
	hipMemcpyToSymbol(HIP_SYMBOL(rPlainSpaceTotal), &uiVal64, sizeof(uiVal64));

	int grSizeX = (task->idxCount-1)/BLOCK_X_SIZE + 1;
	int grSizeY = (grSizeX-1)/GRID_X_SIZE + 1;
	grSizeX = GRID_X_SIZE;
	dim3 numBlocks(grSizeX, grSizeY);
	cuErr = hipSuccess;
	tStart = time(NULL);
	for(int idx = 0; idx < task->rainbowChainLen-1 && cuErr == hipSuccess; idx+=KERN_CHAIN_SIZE) {
		RTGenMD5Kernel<<<numBlocks, BLOCK_X_SIZE>>>(idx, min(idx+KERN_CHAIN_SIZE, task->rainbowChainLen-1));
		cuErr = hipGetLastError();
		if(cuErr == hipSuccess)
			cuErr = hipDeviceSynchronize();
		
	}
	tEnd = time(NULL);
	fprintf(stderr, "Kernel run time: %i\n", (tEnd - tStart));

	if(cuErr == hipSuccess)
		hipMemcpy(resultBuff, data, task->idxCount*2*sizeof(unsigned __int64), hipMemcpyDeviceToHost);
	else
		fprintf(stderr, "Error happened: %d (%s)\n", cuErr, hipGetErrorString(cuErr));	

	hipFree(cpPos);
	hipFree(charSet);
	hipFree(dimVec);
	hipFree(stPlain);
	hipFree(data);
	return cuErr==hipSuccess? task->idxCount : -1;
}

extern "C" int GetChainsBufferSize(int minSize) {
	int grSizeX = (minSize-1)/BLOCK_X_SIZE + 1;
	int grSizeY = (grSizeX-1)/GRID_X_SIZE + 1;
	grSizeX = GRID_X_SIZE;
	return grSizeX*grSizeY*BLOCK_X_SIZE;
}
