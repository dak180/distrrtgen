#include "hip/hip_runtime.h"
// freerainbowtables is a project for generating, distributing, and using
// perfect rainbow tables
//
// Copyright 2010, 2011 Jan Kyska
// Copyright 2010 Martin Westergaard Jørgensen <martinwj2005@gmail.com>
// Copyright 2010, 2011 James Nobis <quel@quelrod.net>
//
// This file is part of freerainbowtables.
//
// freerainbowtables is free software: you can redistribute it and/or modify
// it under the terms of the GNU General Public License as published by
// the Free Software Foundation, either version 2 of the License, or
// (at your option) any later version.
//
// freerainbowtables is distributed in the hope that it will be useful,
// but WITHOUT ANY WARRANTY; without even the implied warranty of
// MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
// GNU General Public License for more details.
//
// You should have received a copy of the GNU General Public License
// along with freerainbowtables.  If not, see <http://www.gnu.org/licenses/>.

#include <stdio.h>  
#include <hip/hip_runtime.h>  
#include "rcuda.h"

#define   GRID_X_L2     6
#define   GRID_Y_L2     6
#define   BLOCK_X_L2    6
#define   GRID_X_SIZE   (1<<GRID_X_L2)
#define   GRID_Y_SIZE   (1<<GRID_Y_L2)
#define   BLOCK_X_SIZE  (1<<BLOCK_X_L2)
#define   PLAIN_MAX_SIZE     20
//#define   KERN_CHAIN_SIZE   100
#define   CHAR_SET_MAXLEN   512
#define   SHIDX(x)      ((x)<<4)


__device__ uint64 *dataHeap;
__constant__ __device__ int *plCpPos;
__constant__ __device__ int plCpPosSize;
__constant__ __device__ uint64 reduceOffset;
__constant__ __device__ uint64 plainSpaceTotal;
__constant__ __device__ uint64 rPlainSpaceTotal;
__constant__ __device__ unsigned char cplChrSet[CHAR_SET_MAXLEN];
__constant__ __device__ uint2 cplDimVec[PLAIN_MAX_SIZE];
__constant__ __device__ uint64 cplStartIdx;
__constant__ __device__ unsigned int cplTargetHash[8];
__constant__ __device__ int cplTargetHashSize;
__device__ int termKernel;

#define RTGEN_PROLOGUE  \
	unsigned int *hData;  \
	unsigned int idx;  \
	uint64 idx64;  \
	unsigned int nPos, ii;  \
	unsigned int cpcheck, checkpoint;  \
	uint3 dimItem;  \
	uint64 uiDiv64, uiVal64, uiMul64;  \
	uint64 plainSpace, idx64mod;  \
	unsigned int uiVal, uiDiv;  \
	unsigned int dimVecIdx;  \
	\
	__shared__ unsigned int shData[SHIDX(BLOCK_X_SIZE)];  \
	\
	if(threadIdx.x == 0) {  \
		nPos = ((((blockIdx.y<<GRID_X_L2) + blockIdx.x)<<BLOCK_X_L2) + threadIdx.x) << 1;  \
		for(ii = 0; ii < BLOCK_X_SIZE; ii++, nPos+=2) {  \
			hData = shData + ((ii>>4)<<8)+(ii&15);  \
			hData[SHIDX(0)] = dataHeap[nPos];  \
			hData[SHIDX(1)] = dataHeap[nPos]>>32;  \
			hData[SHIDX(2)] = dataHeap[nPos+1];  \
		}  \
	}  \
	__syncthreads();  \
	\
	hData = shData + ((threadIdx.x>>4)<<8)+(threadIdx.x&15);  \
	\
	idx64 = hData[SHIDX(1)];  \
	idx64 = (idx64<<32) | hData[SHIDX(0)];  \
	cpcheck = hData[SHIDX(2)];  \
	checkpoint = cpcheck&0x0000ffff;  \
	cpcheck = cpcheck>>16;  \
	\
	for(nPos = chainStart; nPos < chainStop; nPos++) {

#define RTGEN_EPILOGUE  \
		idx64 = hData[SHIDX(1)];  \
		idx64 = (idx64<<32) | hData[SHIDX(0)];  \
		idx64 += reduceOffset + nPos;  \
		uiDiv64 = __umul64hi(idx64, rPlainSpaceTotal);  \
		idx64 -= uiDiv64*plainSpaceTotal;  \
		if(idx64 >= plainSpaceTotal)  \
			idx64 -= plainSpaceTotal;  \
		\
		if(cpcheck < plCpPosSize && nPos == plCpPos[cpcheck]) {  \
			checkpoint |= ((unsigned int)idx64&1) << cpcheck;  \
			cpcheck++;  \
		}  \
	}  \
	\
	hData[SHIDX(0)] = idx64;  \
	hData[SHIDX(1)] = idx64>>32;  \
	hData[SHIDX(2)] = (cpcheck<<16)|(checkpoint&0x0000ffff);  \
	__syncthreads();  \
	\
	if(threadIdx.x == 0) {  \
		nPos = ((((blockIdx.y<<GRID_X_L2) + blockIdx.x)<<BLOCK_X_L2) + threadIdx.x) << 1;  \
		for(ii = 0; ii < BLOCK_X_SIZE; ii++, nPos+=2) {  \
			hData = shData + ((ii>>4)<<8)+(ii&15);  \
			dataHeap[nPos] = ((uint64)hData[SHIDX(1)]<<32)|(uint64)hData[SHIDX(0)];  \
			dataHeap[nPos+1] = hData[SHIDX(2)];  \
		}  \
	}

#define RTGEN_IDX2PLAIN_BEGIN  \
	dimVecIdx = 0;  \
	ii = 0;  \
	do {  \
		plainSpace = (uint64)cplDimVec[dimVecIdx].x | ((uint64)cplDimVec[dimVecIdx].y<<32);  \
		dimVecIdx++;  \
		uiVal64 = idx64/plainSpace;  \
		idx64mod = idx64 - uiVal64*plainSpace;  \
		idx64 = uiVal64

#define RTGEN_IDX2PLAIN_END  \
		if(!(dimItem.z&512)) {  \
			while(!((cplDimVec[dimVecIdx].x>>16)&512))  \
				dimVecIdx++;  \
			dimVecIdx++;  \
		}  \
		\
		idx64 = idx64mod;  \
	} while(plainSpace > 1)

#define RTGEN_I2P_LOOP64_BEGIN  \
	dimItem.z = 0;  \
	for(; idx64 > 0xfffffff0ull && ii < PLAIN_MAX_SIZE && !(dimItem.z&512); ii++, dimVecIdx++) {  \
		uiVal64 = idx64;  \
		dimItem.x = (cplDimVec[dimVecIdx].x&0xffffu);  \
		dimItem.y = cplDimVec[dimVecIdx].y;  \
		dimItem.z = (cplDimVec[dimVecIdx].x>>16);  \
		\
		uiMul64 = (uint64)dimItem.y<<32;  \
		idx64 = __umul64hi(uiVal64, uiMul64);  \
		uiDiv64 = uiVal64 - idx64*(uint64)dimItem.x;  \
		uiVal = __umulhi((unsigned int)uiDiv64, dimItem.y);  \
		uiDiv = (unsigned int)uiDiv64 - uiVal * dimItem.x;  \
		idx64 += uiVal;  \
		if(uiDiv >= dimItem.x) {  \
			uiDiv -= dimItem.x;  \
			idx64++;  \
		}  \
		\
		if(dimItem.z&256) {  \
			if(!uiDiv && idx64>0) {  \
				uiDiv = dimItem.x;  \
				idx64--;  \
			}  \
			uiDiv -= (uiDiv>0?1u:0);  \
		}
		
#define RTGEN_I2P_LOOP64_END  \
	}

#define RTGEN_I2P_LOOP32_BEGIN  \
	dimItem.z = 0;  \
	for(idx = (unsigned int)idx64; ii < PLAIN_MAX_SIZE && !(dimItem.z&512); ii++, dimVecIdx++) {  \
		uiVal = idx;  \
		dimItem.x = (cplDimVec[dimVecIdx].x&0xffffu);  \
		dimItem.y = cplDimVec[dimVecIdx].y;  \
		dimItem.z = (cplDimVec[dimVecIdx].x>>16);  \
		\
		idx = __umulhi(uiVal, dimItem.y);  \
		uiDiv = uiVal - idx*dimItem.x;  \
		if(uiDiv >= dimItem.x) {  \
			uiDiv -= dimItem.x;  \
			idx++;  \
		}  \
		\
		if(dimItem.z&256) {  \
			if(!uiDiv && !idx) {  \
				dimVecIdx++;  \
				break;  \
			}  \
			if(!uiDiv && idx>0) {  \
				uiDiv = dimItem.x;  \
				idx--;  \
			}  \
			uiDiv -= (uiDiv>0?1u:0);  \
		}
			
#define RTGEN_I2P_LOOP32_END  \
	}


/***********   PreCalculate  **************/

#define RTPRECALC_PROLOGUE  \
	unsigned int *hData;  \
	unsigned int idx;  \
	uint64 idx64;  \
	unsigned int nPos, ii;  \
	unsigned int startNPos;  \
	uint3 dimItem;  \
	uint64 uiDiv64, uiVal64, uiMul64;  \
	uint64 plainSpace, idx64mod;  \
	unsigned int uiVal, uiDiv;  \
	unsigned int dimVecIdx;  \
	\
	__shared__ unsigned int shData[SHIDX(BLOCK_X_SIZE)];  \
	\
	if(threadIdx.x == 0) {  \
		nPos = (((blockIdx.y<<GRID_X_L2) + blockIdx.x)<<BLOCK_X_L2) + threadIdx.x;  \
		for(ii = 0; ii < BLOCK_X_SIZE; ii++, nPos++) {  \
			hData = shData + ((ii>>4)<<8)+(ii&15);  \
			hData[SHIDX(0)] = dataHeap[nPos];  \
			hData[SHIDX(1)] = dataHeap[nPos]>>32;  \
			hData[SHIDX(2)] = (unsigned int)cplStartIdx + nPos + 1;  \
		}  \
	}  \
	__syncthreads();  \
	\
	hData = shData + ((threadIdx.x>>4)<<8)+(threadIdx.x&15);  \
	\
	idx64 = hData[SHIDX(1)];  \
	idx64 = (idx64<<32) | hData[SHIDX(0)];  \
	startNPos = hData[SHIDX(2)];  \
	\
	for(nPos = (startNPos>chainStart? startNPos : chainStart); nPos < chainStop; nPos++) {

#define RTPRECALC_EPILOGUE  \
		idx64 = hData[SHIDX(1)];  \
		idx64 = (idx64<<32) | hData[SHIDX(0)];  \
		idx64 += reduceOffset + nPos;  \
		uiDiv64 = __umul64hi(idx64, rPlainSpaceTotal);  \
		idx64 -= uiDiv64*plainSpaceTotal;  \
		if(idx64 >= plainSpaceTotal)  \
			idx64 -= plainSpaceTotal;  \
		\
	}  \
	\
	hData[SHIDX(0)] = idx64;  \
	hData[SHIDX(1)] = idx64>>32;  \
	__syncthreads();  \
	\
	if(threadIdx.x == 0) {  \
		nPos = ((((blockIdx.y<<GRID_X_L2) + blockIdx.x)<<BLOCK_X_L2) + threadIdx.x);  \
		for(ii = 0; ii < BLOCK_X_SIZE; ii++, nPos++) {  \
			hData = shData + ((ii>>4)<<8)+(ii&15);  \
			dataHeap[nPos] = ((uint64)hData[SHIDX(1)]<<32)|(uint64)hData[SHIDX(0)];  \
		}  \
	}


/***********   CheckAlarm  **************/

#define RTCHKALARM_PROLOGUE  \
	unsigned int *hData;  \
	unsigned int idx;  \
	uint64 idx64, lastIdx64;  \
	unsigned int nPos, ii;  \
	unsigned int stopNPos;  \
	uint3 dimItem;  \
	uint64 uiDiv64, uiVal64, uiMul64;  \
	uint64 plainSpace, idx64mod;  \
	unsigned int uiVal, uiDiv;  \
	unsigned int dimVecIdx;  \
	\
	__shared__ unsigned int shData[SHIDX(BLOCK_X_SIZE)];  \
	\
	if(threadIdx.x == 0) {  \
		nPos = (((((blockIdx.y<<GRID_X_L2) + blockIdx.x)<<BLOCK_X_L2) + threadIdx.x) << 1);  \
		for(ii = 0; ii < BLOCK_X_SIZE; ii++, nPos+=2) {  \
			hData = shData + ((ii>>4)<<8)+(ii&15);  \
			hData[SHIDX(0)] = dataHeap[nPos];  \
			hData[SHIDX(1)] = dataHeap[nPos]>>32;  \
			hData[SHIDX(cplTargetHashSize+1)] = dataHeap[nPos+1];  \
		}  \
	}  \
	__syncthreads();  \
	\
	hData = shData + ((threadIdx.x>>4)<<8)+(threadIdx.x&15);  \
	\
	idx64 = hData[SHIDX(1)];  \
	idx64 = (idx64<<32) | hData[SHIDX(0)];  \
	stopNPos = hData[SHIDX(cplTargetHashSize+1)];  \
	if(chainStart > stopNPos)  \
		return; \
	chainStop = (chainStop>stopNPos? stopNPos : chainStop); \
	\
	for(nPos = chainStart; nPos <= chainStop; nPos++) { \
		lastIdx64 = idx64;

#define RTCHKALARM_EPILOGUE  \
		if(nPos == stopNPos)  \
			break;  \
		idx64 = hData[SHIDX(1)];  \
		idx64 = (idx64<<32) | hData[SHIDX(0)];  \
		idx64 += reduceOffset + nPos;  \
		uiDiv64 = __umul64hi(idx64, rPlainSpaceTotal);  \
		idx64 -= uiDiv64*plainSpaceTotal;  \
		if(idx64 >= plainSpaceTotal)  \
			idx64 -= plainSpaceTotal;  \
		\
	}  \
	\
	if(nPos < stopNPos) {  \
		hData[SHIDX(0)] = idx64;  \
		hData[SHIDX(1)] = idx64>>32;  \
		hData[SHIDX(cplTargetHashSize)] = 0;  \
	} else {  \
		uiVal = 1;  \
		for(nPos = 0; nPos < cplTargetHashSize; nPos++)  \
			uiVal = (cplTargetHash[nPos]==hData[SHIDX(nPos)]? uiVal : 0);  \
		hData[SHIDX(0)] = lastIdx64;  \
		hData[SHIDX(1)] = lastIdx64>>32;  \
		hData[SHIDX(cplTargetHashSize)] = uiVal;  \
	}  \
	__syncthreads();  \
	\
	if(threadIdx.x == 0) {  \
		nPos = (((((blockIdx.y<<GRID_X_L2) + blockIdx.x)<<BLOCK_X_L2) + threadIdx.x) << 1);  \
		for(ii = 0; ii < BLOCK_X_SIZE; ii++, nPos+=2) {  \
			hData = shData + ((ii>>4)<<8)+(ii&15);  \
			dataHeap[nPos] = ((uint64)hData[SHIDX(1)]<<32)|(uint64)hData[SHIDX(0)];  \
			if(hData[SHIDX(cplTargetHashSize)]) {  \
				dataHeap[nPos+1] = (1ull<<63);  \
				termKernel = 1;  \
			}  \
		}  \
	}


#include "rcuda_md5.inc"
#include "rcuda_md4.inc"
#include "rcuda_ntlm.inc"
#include "rcuda_sha1.inc"
#include "rcuda_lm.inc"

extern "C" int CalcChainsOnCUDA(const rcuda::RCudaTask* task, uint64 *resultBuff) {
	hipSetDeviceFlags(hipDeviceScheduleBlockingSync);

	hipError_t cuErr;
	uint64 *data;
	int *cpPos;
	uint64 uiVal64;

	if(task->charSetSize > CHAR_SET_MAXLEN)
		return -1;

	switch(task->hash) {
	case rcuda::RHASH_MD5:
	case rcuda::RHASH_MD4:
	case rcuda::RHASH_NTLM:
	case rcuda::RHASH_SHA1:
	case rcuda::RHASH_MYSQLSHA1:
	case rcuda::RHASH_LM:
		break;
	default:	
		return 0;
	}
	
	hipMalloc((void**)&data, (task->idxCount+BLOCK_X_SIZE)*2*sizeof(uint64));
	hipMalloc((void**)&cpPos, task->cpPosSize*sizeof(int));

	hipMemcpy(data, resultBuff, task->idxCount*2*sizeof(uint64), hipMemcpyHostToDevice);
	hipMemcpy(cpPos, task->cpPositions, task->cpPosSize*sizeof(int), hipMemcpyHostToDevice);

	hipMemcpyToSymbol(HIP_SYMBOL(dataHeap), &data, sizeof(data));
	hipMemcpyToSymbol(HIP_SYMBOL(cplDimVec), task->dimVec, min(task->dimVecSize, PLAIN_MAX_SIZE)*sizeof(uint2));
	hipMemcpyToSymbol(HIP_SYMBOL(cplChrSet), task->charSet, min(task->charSetSize, CHAR_SET_MAXLEN));
	hipMemcpyToSymbol(HIP_SYMBOL(plCpPos), &cpPos, sizeof(cpPos));
	hipMemcpyToSymbol(HIP_SYMBOL(plCpPosSize), &task->cpPosSize, sizeof(task->cpPosSize));
	hipMemcpyToSymbol(HIP_SYMBOL(reduceOffset), &task->reduceOffset, sizeof(task->reduceOffset));
	hipMemcpyToSymbol(HIP_SYMBOL(plainSpaceTotal), &task->plainSpaceTotal, sizeof(task->plainSpaceTotal));
	uiVal64 = (uint64)-1/task->plainSpaceTotal;
	hipMemcpyToSymbol(HIP_SYMBOL(rPlainSpaceTotal), &uiVal64, sizeof(uiVal64));

	int grSizeX = (task->idxCount-1)/BLOCK_X_SIZE + 1;
	int grSizeY = (grSizeX-1)/GRID_X_SIZE + 1;
	grSizeX = GRID_X_SIZE;
	dim3 numBlocks(grSizeX, grSizeY);
	cuErr = hipSuccess;

	for(unsigned int idx = 0; idx < task->rainbowChainLen-1 && cuErr == hipSuccess; idx+=task->kernChainSize) {
		switch(task->hash) {
		case rcuda::RHASH_MD5:
			RTGenMD5Kernel<<<numBlocks, BLOCK_X_SIZE>>>(idx, min(idx+task->kernChainSize, task->rainbowChainLen-1));
			break;
		case rcuda::RHASH_MD4:
			RTGenMD4Kernel<<<numBlocks, BLOCK_X_SIZE>>>(idx, min(idx+task->kernChainSize, task->rainbowChainLen-1));
			break;
		case rcuda::RHASH_NTLM:
			RTGenNTLMKernel<<<numBlocks, BLOCK_X_SIZE>>>(idx, min(idx+task->kernChainSize, task->rainbowChainLen-1));
			break;
		case rcuda::RHASH_SHA1:
			RTGenSHA1Kernel<<<numBlocks, BLOCK_X_SIZE>>>(idx, min(idx+task->kernChainSize, task->rainbowChainLen-1));
			break;
		case rcuda::RHASH_MYSQLSHA1:
			RTGenMySQLSHA1Kernel<<<numBlocks, BLOCK_X_SIZE>>>(idx, min(idx+task->kernChainSize, task->rainbowChainLen-1));
			break;
		case rcuda::RHASH_LM:
			RTGenLMKernel<<<numBlocks, BLOCK_X_SIZE>>>(idx, min(idx+task->kernChainSize, task->rainbowChainLen-1));
			break;
		case rcuda::RHASH_UNDEF:
			break;
		}

		cuErr = hipDeviceSynchronize();
	}

	if(cuErr == hipSuccess)
		hipMemcpy(resultBuff, data, task->idxCount*2*sizeof(uint64), hipMemcpyDeviceToHost);
	else
		fprintf(stderr, "Error happened: %d (%s)\n", cuErr, hipGetErrorString(cuErr));	

	hipFree(cpPos);
	hipFree(data);
	return cuErr==hipSuccess? task->idxCount : -1;
}

extern "C" int PreCalculateOnCUDA(const rcuda::RCudaTask* task, uint64 *resultBuff) {
	hipSetDeviceFlags(hipDeviceScheduleBlockingSync);

	hipError_t cuErr;
	uint64 *data;
	uint64 uiVal64;

	if(task->charSetSize > CHAR_SET_MAXLEN)
		return -1;

	switch(task->hash) {
	case rcuda::RHASH_MD5:
	case rcuda::RHASH_MD4:
	case rcuda::RHASH_NTLM:
	case rcuda::RHASH_SHA1:
	case rcuda::RHASH_MYSQLSHA1:
	case rcuda::RHASH_LM:
		break;
	default:	
		return 0;
	}

	hipMalloc((void**)&data, (task->idxCount+BLOCK_X_SIZE)*sizeof(uint64));
	hipMemcpy(data, resultBuff, task->idxCount*sizeof(uint64), hipMemcpyHostToDevice);

	hipMemcpyToSymbol(HIP_SYMBOL(dataHeap), &data, sizeof(data));
	hipMemcpyToSymbol(HIP_SYMBOL(cplDimVec), task->dimVec, min(task->dimVecSize, PLAIN_MAX_SIZE)*sizeof(uint2));
	hipMemcpyToSymbol(HIP_SYMBOL(cplChrSet), task->charSet, min(task->charSetSize, CHAR_SET_MAXLEN));
	hipMemcpyToSymbol(HIP_SYMBOL(reduceOffset), &task->reduceOffset, sizeof(task->reduceOffset));
	hipMemcpyToSymbol(HIP_SYMBOL(plainSpaceTotal), &task->plainSpaceTotal, sizeof(task->plainSpaceTotal));
	uiVal64 = (uint64)-1/task->plainSpaceTotal;
	hipMemcpyToSymbol(HIP_SYMBOL(rPlainSpaceTotal), &uiVal64, sizeof(uiVal64));
	hipMemcpyToSymbol(HIP_SYMBOL(cplStartIdx), &task->startIdx, sizeof(task->startIdx));
	
	int grSizeX = (task->idxCount-1)/BLOCK_X_SIZE + 1;
	int grSizeY = (grSizeX-1)/GRID_X_SIZE + 1;
	grSizeX = GRID_X_SIZE;
	dim3 numBlocks(grSizeX, grSizeY);
	cuErr = hipSuccess;

	for(unsigned int idx = (unsigned int)task->startIdx; idx < task->rainbowChainLen-1 && cuErr == hipSuccess; idx+=task->kernChainSize) {
		switch(task->hash) {
		case rcuda::RHASH_MD5:
			RTPreCalcMD5Kernel<<<numBlocks, BLOCK_X_SIZE>>>(idx, min(idx+task->kernChainSize, task->rainbowChainLen-1));
			break;
		case rcuda::RHASH_MD4:
			RTPreCalcMD4Kernel<<<numBlocks, BLOCK_X_SIZE>>>(idx, min(idx+task->kernChainSize, task->rainbowChainLen-1));
			break;
		case rcuda::RHASH_NTLM:
			RTPreCalcNTLMKernel<<<numBlocks, BLOCK_X_SIZE>>>(idx, min(idx+task->kernChainSize, task->rainbowChainLen-1));
			break;
		case rcuda::RHASH_SHA1:
			RTPreCalcSHA1Kernel<<<numBlocks, BLOCK_X_SIZE>>>(idx, min(idx+task->kernChainSize, task->rainbowChainLen-1));
			break;
		case rcuda::RHASH_MYSQLSHA1:
			RTPreCalcMySQLSHA1Kernel<<<numBlocks, BLOCK_X_SIZE>>>(idx, min(idx+task->kernChainSize, task->rainbowChainLen-1));
			break;
		case rcuda::RHASH_LM:
			RTPreCalcKernel<<<numBlocks, BLOCK_X_SIZE>>>(idx, min(idx+task->kernChainSize, task->rainbowChainLen-1));
			break;
		case rcuda::RHASH_UNDEF:
			break;
		}

		cuErr = hipDeviceSynchronize();
	}

	if(cuErr == hipSuccess)
		hipMemcpy(resultBuff, data, task->idxCount*sizeof(uint64), hipMemcpyDeviceToHost);
	else
		fprintf(stderr, "Error happened: %d (%s)\n", cuErr, hipGetErrorString(cuErr));	

	hipFree(data);
	return cuErr==hipSuccess? task->idxCount : -1;
}

extern "C" int CheckAlarmOnCUDA(const rcuda::RCudaTask* task, uint64 *resultBuff) {
	hipSetDeviceFlags(hipDeviceScheduleBlockingSync);

	hipError_t cuErr;
	uint64 *data;
	uint64 uiVal64;
	int isize;

	if(task->charSetSize > CHAR_SET_MAXLEN)
		return -1;

	isize = 0;
	switch(task->hash) {
	case rcuda::RHASH_MD5:
	case rcuda::RHASH_MD4:
	case rcuda::RHASH_NTLM:
		isize = (isize==0? 16 : isize);
	case rcuda::RHASH_SHA1:
	case rcuda::RHASH_MYSQLSHA1:
		isize = (isize==0? 20 : isize);
	case rcuda::RHASH_LM:
		isize = (isize==0? 8 : isize);
		hipMemcpyToSymbol(HIP_SYMBOL(cplTargetHash), task->targetHash, isize);
		isize = (isize>>2);
		hipMemcpyToSymbol(HIP_SYMBOL(cplTargetHashSize), &isize, sizeof(isize));
		break;
	default:	
		return 0;
	}

	hipMalloc((void**)&data, (task->idxCount+BLOCK_X_SIZE)*2*sizeof(uint64));
	hipMemcpy(data, resultBuff, task->idxCount*2*sizeof(uint64), hipMemcpyHostToDevice);

	hipMemcpyToSymbol(HIP_SYMBOL(dataHeap), &data, sizeof(data));
	hipMemcpyToSymbol(HIP_SYMBOL(cplDimVec), task->dimVec, min(task->dimVecSize, PLAIN_MAX_SIZE)*sizeof(uint2));
	hipMemcpyToSymbol(HIP_SYMBOL(cplChrSet), task->charSet, min(task->charSetSize, CHAR_SET_MAXLEN));
	hipMemcpyToSymbol(HIP_SYMBOL(reduceOffset), &task->reduceOffset, sizeof(task->reduceOffset));
	hipMemcpyToSymbol(HIP_SYMBOL(plainSpaceTotal), &task->plainSpaceTotal, sizeof(task->plainSpaceTotal));
	uiVal64 = (uint64)-1/task->plainSpaceTotal;
	hipMemcpyToSymbol(HIP_SYMBOL(rPlainSpaceTotal), &uiVal64, sizeof(uiVal64));
	
	int grSizeX = (task->idxCount-1)/BLOCK_X_SIZE + 1;
	int grSizeY = (grSizeX-1)/GRID_X_SIZE + 1;
	grSizeX = GRID_X_SIZE;
	dim3 numBlocks(grSizeX, grSizeY);
	cuErr = hipSuccess;

	isize = 0;
	hipMemcpyToSymbol(HIP_SYMBOL(termKernel), &isize, sizeof(isize));
	for(unsigned int idx = 0; idx <= task->rainbowChainLen && cuErr == hipSuccess; idx+=task->kernChainSize) {
		switch(task->hash) {
		case rcuda::RHASH_MD5:
			RTCheckAlarmMD5Kernel<<<numBlocks, BLOCK_X_SIZE>>>(idx, min(idx+task->kernChainSize-1, task->rainbowChainLen));
			break;
		case rcuda::RHASH_MD4:
			RTCheckAlarmMD4Kernel<<<numBlocks, BLOCK_X_SIZE>>>(idx, min(idx+task->kernChainSize-1, task->rainbowChainLen));
			break;
		case rcuda::RHASH_NTLM:
			RTCheckAlarmNTLMKernel<<<numBlocks, BLOCK_X_SIZE>>>(idx, min(idx+task->kernChainSize-1, task->rainbowChainLen));
			break;
		case rcuda::RHASH_SHA1:
			RTCheckAlarmSHA1Kernel<<<numBlocks, BLOCK_X_SIZE>>>(idx, min(idx+task->kernChainSize-1, task->rainbowChainLen));
			break;
		case rcuda::RHASH_MYSQLSHA1:
			RTCheckAlarmMySQLSHA1Kernel<<<numBlocks, BLOCK_X_SIZE>>>(idx, min(idx+task->kernChainSize-1, task->rainbowChainLen));
			break;
		case rcuda::RHASH_LM:
			RTCheckAlarmLMKernel<<<numBlocks, BLOCK_X_SIZE>>>(idx, min(idx+task->kernChainSize-1, task->rainbowChainLen));
			break;
		case rcuda::RHASH_UNDEF:
			break;
		}

		cuErr = hipDeviceSynchronize();
		hipMemcpyFromSymbol(&isize, HIP_SYMBOL(termKernel), sizeof(isize));
		if(isize) 
			break;
	}

	if(cuErr == hipSuccess)
		hipMemcpy(resultBuff, data, task->idxCount*2*sizeof(uint64), hipMemcpyDeviceToHost);
	else
		fprintf(stderr, "Error happened: %d (%s)\n", cuErr, hipGetErrorString(cuErr));	

	hipFree(data);
	return cuErr==hipSuccess? task->idxCount : -1;
}

extern "C" int GetChainsBufferSize(int minSize) {
	int grSizeX = (minSize-1)/BLOCK_X_SIZE + 1;
	int grSizeY = (grSizeX-1)/GRID_X_SIZE + 1;
	grSizeX = GRID_X_SIZE;
	return grSizeX*grSizeY*BLOCK_X_SIZE;
}

extern "C" int SetCudaDevice(int device) {
	return hipSetDevice(device);
}
